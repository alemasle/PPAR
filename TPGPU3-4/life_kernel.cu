
#include <hip/hip_runtime.h>

__device__ int read_cell(int * source_domain, int x, int y, int dx, int dy, unsigned int domain_x, unsigned int domain_y){
  x = (unsigned int)(x + dx + domain_x) % domain_x;
  y = (unsigned int)(y + dy + domain_y) % domain_y;
  return source_domain[y * domain_x + x];
}

__device__ void write_cell(int * source_domain, int x, int y, int dx, int dy, unsigned int domain_x, unsigned int domain_y, int val){
  x = (unsigned int)(x + dx + domain_x) % domain_x;
  y = (unsigned int)(y + dy + domain_y) % domain_y;
  source_domain[y * domain_x + x] = val;
}

// 1 cell per thread, 1 cell per word kernel
__global__ void life_kernel(int * source_domain, int * dest_domain, int domain_x, int domain_y) {

  extern __shared__ int cells[];

  int tx = blockIdx.x * (blockDim.x-2) + threadIdx.x-1;
  int ty = blockIdx.y * (blockDim.y-2) + threadIdx.y-1;

  int myself = read_cell(source_domain,tx,ty,0,0,domain_x,domain_y);

  cells[threadIdx.y*blockDim.x+threadIdx.x] = myself;

  __syncthreads();

  // Read the 8 neighbors and count number of blue and red
  int num_red = 0;
  int num_blue = 0;

  int neighbors[8];

  neighbors[0] = read_cell(source_domain, tx,ty, -1,-1, domain_x, domain_y);
  neighbors[1] = read_cell(source_domain, tx,ty, -1, 1, domain_x, domain_y);
  neighbors[2] = read_cell(source_domain, tx,ty,  1,-1, domain_x, domain_y);
  neighbors[3] = read_cell(source_domain, tx,ty,  1, 1, domain_x, domain_y);
  neighbors[4] = read_cell(source_domain, tx,ty,  0,-1, domain_x, domain_y);
  neighbors[5] = read_cell(source_domain, tx,ty,  0, 1, domain_x, domain_y);
  neighbors[6] = read_cell(source_domain, tx,ty,  1, 0, domain_x, domain_y);
  neighbors[7] = read_cell(source_domain, tx,ty, -1, 0, domain_x, domain_y);

  for(int i = 0; i < 8; i++){
    if(neighbors[i] == 1){
      num_red ++;
    }
    else if(neighbors[i] == 2){
      num_blue ++;
    }
  }

  int tot = num_red + num_blue;
  int newVal = myself;

  if(myself == 0 && tot == 3){  // If it born
    newVal = num_blue > num_red ? 2 : 1;
  }

  if(myself != 0 && (tot > 3 || tot < 2)){  // If it dies
    newVal = 0;
  }

  else if(myself != 0 && (tot <= 3 || tot >= 2)){ // He continues to live
    newVal = myself;
  }

  write_cell(source_domain, tx, ty, 0, 0, domain_x, domain_y, newVal);
}
